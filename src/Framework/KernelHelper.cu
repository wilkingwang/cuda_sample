#include <hip/hip_runtime.h>

#include "HelpCuda.h"
#include "KernelHelper.cuh"

void allocateArray(void** devPtr, size_t size)
{
	checkCudaErrors(hipMalloc(devPtr, size));
}

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
	checkCudaErrors(hipMemcpy((char*)device + offset, host, size, hipMemcpyHostToDevice));
}

void copyArrayFromDevice(void* host, const void* device, int size)
{
	checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
}